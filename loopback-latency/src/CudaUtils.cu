#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <hip/hip_runtime.h>

#include "CudaUtils.h"
#include "Console.h"

void* CudaAlloc(size_t size, bool enableRDMA)
{
    void* ptr;
    hipMalloc(&ptr, size);
    if (enableRDMA)
    {
        unsigned int syncFlag = 1;
        if (hipPointerSetAttribute(&syncFlag, HIP_POINTER_ATTRIBUTE_SYNC_MEMOPS, (hipDeviceptr_t)ptr))
        {
            Error("Failed to set HIP_POINTER_ATTRIBUTE_SYNC_MEMOPS");
            hipFree(ptr);
            return nullptr;
        }
    }
    return ptr;
}

void CudaFree(void* ptr)
{
    hipFree(ptr);
}

void CudaMemcpyDtoH(void* host, void* dev, size_t bytes)
{
    hipMemcpy(host, dev, bytes, hipMemcpyDeviceToHost);
}

void CudaMemcpyHtoD(void* dev, void* host, size_t bytes)
{
    hipMemcpy(dev, host, bytes, hipMemcpyHostToDevice);
}

__global__
void WriteRGBA(uint32_t *ptr, size_t elementCount, uint32_t value)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < elementCount; i += stride)
    {
        ptr[i] = value;
    }
}

void CudaWriteRGBA(uint32_t* ptr, size_t elementCount, uint8_t r, uint8_t g, uint8_t b)
{
    unsigned int blockSize = 1024;
    unsigned int numBlocks = (elementCount + blockSize - 1) / blockSize;

    uint32_t abgr = (0xFF << 24) | (b << 16) | (g << 8) | (r << 0);
    WriteRGBA<<<numBlocks, blockSize>>>(ptr, elementCount, abgr);

    hipStreamSynchronize(hipStreamPerThread);
}

__global__
void SimulateProcessing(uint32_t* ptr, size_t elementCount, size_t loopCount)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < elementCount; i += stride)
    {
        int val = 0;
        for (int j = 0; j < loopCount; j++)
        {
            val += sin((i + j) / 1000.0f);
        }
        ptr[i] = val;
    }
}

void CudaSimulateProcessing(uint32_t* ptr, size_t elementCount, size_t loopCount)
{
    if (!loopCount)
        return;

    unsigned int blockSize = 1024;
    unsigned int numBlocks = (elementCount + blockSize - 1) / blockSize;

    SimulateProcessing<<<numBlocks, blockSize>>>(ptr, elementCount, loopCount);

    hipStreamSynchronize(hipStreamPerThread);
}
